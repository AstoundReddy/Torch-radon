#include "texture.h"
#include <iostream>

TextureCache::TextureCache(){}

void TextureCache::free(){
    //std::cout << "Free" << std::endl;
    if(this->array != nullptr){
        checkCudaErrors(hipFreeArray(this->array));
        checkCudaErrors(hipDestroyTextureObject(this->texObj));
    }
}

void TextureCache::allocate(uint b, uint w, uint h){
    // free previously allocated array
    this->free();

    this->batch_size = b;
    this->width = w;
    this->height = h;

    // Allocate a layered CUDA array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    const hipExtent extent = make_hipExtent(width, height, batch_size);
    checkCudaErrors(hipMalloc3DArray(&array, &channelDesc, extent, hipArrayLayered));

    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = array;

    // Specify texture object parameters
    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeBorder;
    texDesc.addressMode[1] = hipAddressModeBorder;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // Create texture object
    checkCudaErrors(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL));
}

void TextureCache::put(const float *data, uint b, uint w, uint h, uint pitch){
    // only reallocate when required
    if(this->batch_size != b || this->width != w ||  this->height != h){
        //std::cout << "Alloc" << std::endl;
        std::cout << this->batch_size << " " << b << " " <<  this->width  << " " <<  w  << " " <<  this->height  << " " <<  h << std::endl;
        //this->allocate(b, w, h);
    }

    // copy data into array
    hipMemcpy3DParms myparms = {0};
    myparms.srcPos = make_hipPos(0, 0, 0);
    myparms.dstPos = make_hipPos(0, 0, 0);
    myparms.srcPtr = make_hipPitchedPtr((void *) data, pitch * sizeof(float), width, height);
    myparms.dstArray = this->array;
    myparms.extent = make_hipExtent(width, height, batch_size);
    myparms.kind = hipMemcpyDeviceToDevice;
    checkCudaErrors(hipMemcpy3D(&myparms));
}
