#include "hip/hip_runtime.h"
#include "radon_noise.h"
#include <hiprand/hiprand_kernel.h>


__global__ void initialize_random_states(hiprandState *state, const uint seed){
    const uint sequence_id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, sequence_id, 0, &state[sequence_id]);
}

__global__ void radon_sinogram_noise(float* sinogram, hiprandState *state, const float sino_max, const float signal, const uint width, const uint height){
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;
    const uint tid = y * blockDim.x * gridDim.x + x;
    const uint y_step = blockDim.y * gridDim.y;

    // load hiprand state in local memory
    hiprandState localState = state[tid];

    // loop through down the sinogram adding noise
    for(uint yy = y; yy < height; yy += y_step){
        uint pos = yy * width + x;
        float reading = hiprand_poisson(&localState, signal * exp(-sinogram[pos]/sino_max));
        sinogram[pos] = -sino_max * log(reading / signal);
    }

    // save hiprand state back in global memory
    state[tid] = localState;
}
