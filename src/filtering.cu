#include "hip/hip_runtime.h"
#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#include "utils.h"


__global__ void apply_filter(hipfftComplex *sino, const int fft_size, const float scaling) {
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < fft_size) {
        sino[fft_size * y + x].x *= float(x) / scaling;
        sino[fft_size * y + x].y *= float(x) / scaling;
    }
}


void radon_filter_sinogram_cuda(const float *x, float *y, const int batch_size, const int n_angles, const int n_rays) {
    const int rows = batch_size * n_angles;
    const int padded_size = next_power_of_two(n_rays * 2);
    // cuFFT only stores half of the coefficient because they are symmetric (see cuFFT documentation)
    const int fft_size = padded_size / 2 + 1;

    // pad x
    hipfftReal *padded_data = nullptr;
    checkCudaErrors(hipMalloc((void **) &padded_data, sizeof(hipfftReal) * rows * padded_size));
    checkCudaErrors(hipMemset(padded_data, 0, sizeof(hipfftReal) * rows * padded_size));
    checkCudaErrors(hipMemcpy2D(padded_data, sizeof(hipfftReal) * padded_size, x, sizeof(float) * n_rays,
                                 sizeof(float) * n_rays, rows, hipMemcpyDeviceToDevice));

    // allocate complex tensor to store FFT coefficients
    hipfftComplex *complex_data = nullptr;
    checkCudaErrors(hipMalloc((void **) &complex_data, sizeof(hipfftComplex) * rows * fft_size));

    // allocate real tensor to store padded filtered sinogram
    hipfftReal *filtered_padded_sino = nullptr;
    checkCudaErrors(hipMalloc((void **) &filtered_padded_sino, sizeof(hipfftReal) * rows * padded_size));
    checkCudaErrors(hipMemset(filtered_padded_sino, 0, sizeof(hipfftReal) * rows * padded_size));

    // create plans for FFT and iFFT
    hipfftHandle forward_plan, back_plan;
    cufftSafeCall(hipfftPlan1d(&forward_plan, padded_size, HIPFFT_R2C, rows));
    cufftSafeCall(hipfftPlan1d(&back_plan, padded_size, HIPFFT_C2R, rows));

    // do FFT
    cufftSafeCall(hipfftExecR2C(forward_plan, padded_data, complex_data));

    // filter in Fourier domain
    apply_filter << < dim3(fft_size / 16 + 1, rows / 16), dim3(16, 16) >> > (complex_data, fft_size, padded_size*padded_size);

    // do iFFT
    cufftSafeCall(hipfftExecC2R(back_plan, complex_data, filtered_padded_sino));

    // copy unpadded result in y
    checkCudaErrors(hipMemcpy2D(y, sizeof(float) * n_rays, filtered_padded_sino, sizeof(float) * padded_size,
                                 sizeof(float) * n_rays, rows, hipMemcpyDeviceToDevice));

    cufftSafeCall(hipfftDestroy(forward_plan));
    cufftSafeCall(hipfftDestroy(back_plan));
}