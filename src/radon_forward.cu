#include "hip/hip_runtime.h"
#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#include "utils.h"
#include "texture.h"

__global__ void radon_forward_kernel(float* __restrict__ output, hipTextureObject_t texObj, const float* __restrict__ rays, const float* __restrict__ angles,
                                     const int img_size, const int n_rays, const int n_angles) {
    // Calculate texture coordinates
    const uint ray_id = blockIdx.x * blockDim.x + threadIdx.x;
    const uint angle_id = blockIdx.y * blockDim.y + threadIdx.y;
    const uint batch_id = blockIdx.z;

    const float rsx = rays[ray_id * 4 + 0];
    const float rsy = rays[ray_id * 4 + 1];
    const float rex = rays[ray_id * 4 + 2];
    const float rey = rays[ray_id * 4 + 3];
    const float v = img_size / 2; //

    const uint n_steps = __float2uint_ru(hypot(rex - rsx, rey - rsy)) + 1; //
    const float vx = (rex - rsx) / n_steps; //
    const float vy = (rey - rsy) / n_steps; //
    const float n = hypot(vx, vy); //

    // rotate ray
    float angle = angles[angle_id];
    float cs = __cosf(angle);
    float sn = __sinf(angle);

    float sx = rsx * cs - rsy * sn + v;
    float sy = rsx * sn + rsy * cs + v;
    float rvx = vx * cs - vy * sn;
    float rvy = vx * sn + vy * cs;

    float tmp = 0.0;
    for (uint j = 0; j < n_steps; j++) {
        tmp += tex2DLayered<float>(texObj, sx + rvx * j, sy + rvy * j, batch_id);
    }

    output[batch_id * n_rays * n_angles + angle_id * n_rays + ray_id] = tmp * n;
}

__global__ void radon_backward_kernel(float *output, hipTextureObject_t texObj, const float *rays, const float *angles,
                                      const int img_size, const int n_rays, const int n_angles) {

    __shared__ float s_sin[256];
    __shared__ float s_cos[256];

    // Calculate image coordinates
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;
    const uint batch_id = blockIdx.z;
    const uint tid = threadIdx.y * blockDim.x + threadIdx.x;

    if(tid < n_angles){
        s_sin[tid] = __sinf(angles[tid]);
        s_cos[tid] = __cosf(angles[tid]);
    }
    __syncthreads();

    const float v = img_size / 2;
    const float dx = (float) x - v + 0.5;
    const float dy = (float) y - v + 0.5;

    float tmp = 0.0;
    const float r = hypot(dx, dy);

    if(r <= 64){
        for (int i = 0; i < n_angles; i++) {
            float j = s_cos[i] * dx + s_sin[i] * dy + v;
            tmp += tex2DLayered<float>(texObj, j, i + 0.5f, batch_id);
        }
    }

    output[batch_id * img_size * img_size + y * img_size + x] = tmp;
}

__global__ void initialize_random_states(hiprandState *state, const uint seed){
    const uint sequence_id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, sequence_id, 0, &state[sequence_id]);
}

__global__ void radon_sinogram_noise(float* sinogram, hiprandState *state, const float sino_max, const float signal, const uint width, const uint height){
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;
    const uint tid = y * blockDim.x * gridDim.x + x;
    const uint y_step = blockDim.y * gridDim.y;

    // load hiprand state in local memory
    hiprandState localState = state[tid];

    // loop through down the sinogram adding noise
    for(uint yy = y; yy < height; yy += y_step){
        uint pos = yy * width + x;
        float reading = hiprand_poisson(&localState, signal * exp(-sinogram[pos]/sino_max));
        sinogram[pos] = -sino_max * log(reading / signal);
    }

    // save hiprand state back in global memory
    state[tid] = localState;
}


void radon_forward_cuda(const float *x, const float *rays, const float *angles, float *y, TextureCache tex_cache, const int batch_size,
                        const int img_size, const int n_rays, const int n_angles) {
    // copy x into CUDA Array (allocating it if needed) and bind to texture
    tex_cache.put(x, batch_size, img_size, img_size, img_size);

    // Invoke kernel
    const int grid_size = img_size / 16;
    dim3 dimGrid(grid_size, n_angles/16, batch_size);
    dim3 dimBlock(16, 16);

    radon_forward_kernel <<< dimGrid, dimBlock >>> (y, tex_cache.texObj, rays, angles, img_size, n_rays, n_angles);
}

void radon_backward_cuda(const float *x, const float *rays, const float *angles, float *y, TextureCache tex_cache, const int batch_size, const int img_size, const int n_rays, const int n_angles) {
    // copy x into CUDA Array (allocating it if needed) and bind to texture
    tex_cache.put(x, batch_size, n_rays, n_angles, n_rays);

    // Invoke kernel
    const int grid_size = img_size / 16;
    dim3 dimGrid(grid_size, grid_size, batch_size);
    dim3 dimBlock(16, 16);

    radon_backward_kernel <<< dimGrid, dimBlock >>> (y, tex_cache.texObj, rays, angles, img_size, n_rays, n_angles);
}

int main() {
    std::cout << "float: " << sizeof(float) << std::endl;
    std::cout << "hipfftReal: " << sizeof(hipfftReal) << std::endl;
    std::cout << "hipfftComplex: " << sizeof(hipfftComplex) << std::endl;
    std::cout << "Hello CUDA" << std::endl;
}


/*
__global__ void apply_filter(hipfftComplex *sino, const int fft_size) {
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < fft_size) {
        sino[fft_size * y + x].x *= float(x);
        sino[fft_size * y + x].y *= float(x);
    }
}


unsigned int next_power_of_two(unsigned int v) {
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;
}

void radon_filter_sinogram_cuda(const float *x, float *y, const int batch_size, const int n_rays, const int n_angles) {
    const int rows = batch_size * n_angles;
    const int padded_size = next_power_of_two(n_rays * 2);
    // cuFFT only stores half of the coefficient because they are symmetric (see cuFFT documentation)
    const int fft_size = padded_size / 2 + 1;
    std::cout << "rows " << rows << std::endl;
    std::cout << "padded_size " << padded_size << std::endl;
    std::cout << "fft_size " << fft_size << std::endl;

    // pad x
    hipfftReal *padded_data = nullptr;
    checkCudaErrors(hipMalloc((void **) &padded_data, sizeof(hipfftReal) * rows * padded_size));
    checkCudaErrors(hipMemset(padded_data, 0, sizeof(hipfftReal) * rows * padded_size));
    checkCudaErrors(hipMemcpy2D(padded_data, sizeof(hipfftReal) * padded_size, x, sizeof(float) * n_rays,
                                 sizeof(float) * n_rays, rows, hipMemcpyDeviceToDevice));

    // allocate complex tensor to store FFT coefficients
    hipfftComplex *complex_data = nullptr;
    checkCudaErrors(hipMalloc((void **) &complex_data, sizeof(hipfftComplex) * rows * fft_size));

    // allocate real tensor to store padded filtered sinogram
    hipfftReal *filtered_padded_sino = nullptr;
    checkCudaErrors(hipMalloc((void **) &filtered_padded_sino, sizeof(hipfftReal) * rows * padded_size));
    checkCudaErrors(hipMemset(filtered_padded_sino, 0, sizeof(hipfftReal) * rows * padded_size));

    // create plans for FFT and iFFT
    hipfftHandle forward_plan, back_plan;
    checkCudaErrors(hipfftPlan1d(&forward_plan, padded_size, HIPFFT_R2C, rows));
    checkCudaErrors(hipfftPlan1d(&back_plan, padded_size, HIPFFT_C2R, rows));

    // do FFT
    checkCudaErrors(hipfftExecR2C(forward_plan, padded_data, complex_data));

    // filter in Fourier domain
    apply_filter << < dim3(fft_size / 16 + 1, rows / 16), dim3(16, 16) >> > (complex_data, fft_size);

    // do iFFT
    checkCudaErrors(hipfftExecC2R(back_plan, complex_data, filtered_padded_sino));

    // copy unpadded result in y
    checkCudaErrors(hipMemcpy2D(y, sizeof(float) * n_rays, filtered_padded_sino, sizeof(float) * padded_size,
                                 sizeof(float) * n_rays, rows, hipMemcpyDeviceToDevice));

    checkCudaErrors(hipfftDestroy(forward_plan));
    checkCudaErrors(hipfftDestroy(back_plan));
}
*/
