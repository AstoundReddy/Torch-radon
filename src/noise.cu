#include "hip/hip_runtime.h"
#include "noise.h"
#include <iostream>

__global__ void initialize_random_states(hiprandState *state, const uint seed){
    const uint sequence_id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, sequence_id, 0, &state[sequence_id]);
}

template<bool approximate> __global__ void radon_sinogram_noise(float* sinogram, hiprandState *state, const float signal, const float density_normalization, const uint width, const uint height){
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;
    const uint tid = y * blockDim.x * gridDim.x + x;
    const uint y_step = blockDim.y * gridDim.y;

    // load hiprand state in local memory
    hiprandState localState = state[tid];

    // loop down the sinogram adding noise
    for(uint yy = y; yy < height; yy += y_step){
        uint pos = yy * width + x;
        // measured signal = signal * exp(-sinogram[pos])
        // then apply poisson noise
        float mu = __expf(signal - sinogram[pos]/density_normalization);
        float reading;
        if(approximate){
            float var = __fsqrt_rn(mu);
            reading = fmaxf(hiprand_normal(&localState)*var + mu, 1.0f);
        }else{
            reading = fmaxf(hiprand_poisson(&localState, mu), 1.0f);
        }

        // convert back to sinogram scale
        sinogram[pos] = fmaxf((signal -__logf(reading)), 0.0f) * density_normalization;
    }

    // save hiprand state back in global memory
    state[tid] = localState;
}

__global__ void radon_emulate_readings(const float* sinogram, int* readings, hiprandState *state, const float signal, const float density_normalization, const uint width, const uint height){
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;
    const uint tid = y * blockDim.x * gridDim.x + x;
    const uint y_step = blockDim.y * gridDim.y;

    // load hiprand state in local memory
    hiprandState localState = state[tid];

    // loop down the sinogram adding noise
    for(uint yy = y; yy < height; yy += y_step){
        uint pos = yy * width + x;
        // measured signal = signal * exp(-sinogram[pos])
        // then apply poisson noise
        float mu = __expf(signal - sinogram[pos]/density_normalization);
        readings[pos] = hiprand_poisson(&localState, mu);
    }

    // save hiprand state back in global memory
    state[tid] = localState;
}

RadonNoiseGenerator::RadonNoiseGenerator(const uint seed){
    // TODO
    checkCudaErrors(hipSetDevice(0));

    // allocate random states
    checkCudaErrors(hipMalloc((void **)&states, 128*1024 * sizeof(hiprandState)));

    this->set_seed(seed);
}

void RadonNoiseGenerator::set_seed(const uint seed){
    // TODO
    checkCudaErrors(hipSetDevice(0));

    initialize_random_states<<<128,1024>>>(states, seed);
}

void RadonNoiseGenerator::add_noise(float* sinogram, const float signal, const float density_normalization, const bool approximate, const uint width, const uint height){
    // TODO
    checkCudaErrors(hipSetDevice(0));

    if(approximate){
        radon_sinogram_noise<true><<<dim3(width/64, 32*1024/width), dim3(64, 4)>>>(sinogram, states, signal, density_normalization, width, height);
    }else{
        radon_sinogram_noise<false><<<dim3(width/64, 32*1024/width), dim3(64, 4)>>>(sinogram, states, signal, density_normalization, width, height);
    }
}

void RadonNoiseGenerator::emulate_readings(const float* sinogram, int* readings, const float signal, const float density_normalization, const uint width, const uint height){
    // TODO
    checkCudaErrors(hipSetDevice(0));

    radon_emulate_readings<<<dim3(width/64, 32*1024/width), dim3(64, 4)>>>(sinogram, readings, states, signal, density_normalization, width, height);
}

void RadonNoiseGenerator::free(){
    if(this->states != nullptr){
        checkCudaErrors(hipFree(this->states));
    }
}

__global__ void lookup_kernel(const int* readings, float *result, const float* lookup_table, const uint lookup_size, const uint width, const uint height){
    // TODO use shared memory
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;
    const uint y_step = blockDim.y * gridDim.y;

    for(uint yy = y; yy < height; yy += y_step){
        uint pos = yy * width + x;
        int index = min(readings[pos], lookup_size-1);
        result[pos] = lookup_table[index];
    }
}

void readings_lookup_cuda(const int* x, float*  y,const float* lookup_table, const uint lookup_size, const uint width, const uint height){
    // TODO
    checkCudaErrors(hipSetDevice(0));

    lookup_kernel<<<dim3(width/64, 32*1024/width), dim3(64, 4)>>>(x, y, lookup_table, lookup_size, width, height);
}
