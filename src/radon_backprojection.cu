#include "hip/hip_runtime.h"
#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#include "utils.h"
#include "texture.h"

__global__ void radon_backward_kernel(float *output, hipTextureObject_t texObj, const float *rays, const float *angles,
                                      const int img_size, const int n_rays, const int n_angles) {

    __shared__ float s_sin[256];
    __shared__ float s_cos[256];

    // Calculate image coordinates
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;
    const uint batch_id = blockIdx.z;
    const uint tid = threadIdx.y * blockDim.x + threadIdx.x;

    if(tid < n_angles){
        s_sin[tid] = __sinf(angles[tid]);
        s_cos[tid] = __cosf(angles[tid]);
    }
    __syncthreads();

    const float v = img_size / 2;
    const float dx = (float) x - v + 0.5;
    const float dy = (float) y - v + 0.5;

    float tmp = 0.0;
    const float r = hypot(dx, dy);

    if(r <= v){
        for (int i = 0; i < n_angles; i++) {
            float j = s_cos[i] * dx + s_sin[i] * dy + v;
            tmp += tex2DLayered<float>(texObj, j, i + 0.5f, batch_id);
        }
    }

    output[batch_id * img_size * img_size + y * img_size + x] = tmp;
}

void radon_backward_cuda(const float *x, const float *rays, const float *angles, float *y, TextureCache& tex_cache, const int batch_size, const int img_size, const int n_rays, const int n_angles) {
    // copy x into CUDA Array (allocating it if needed) and bind to texture
    tex_cache.put(x, batch_size, n_rays, n_angles, n_rays);

    // Invoke kernel
    const int grid_size = img_size / 16;
    dim3 dimGrid(grid_size, grid_size, batch_size);
    dim3 dimBlock(16, 16);

    radon_backward_kernel <<< dimGrid, dimBlock >>> (y, tex_cache.texObj, rays, angles, img_size, n_rays, n_angles);
}

/*
__global__ void apply_filter(hipfftComplex *sino, const int fft_size) {
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < fft_size) {
        sino[fft_size * y + x].x *= float(x);
        sino[fft_size * y + x].y *= float(x);
    }
}


unsigned int next_power_of_two(unsigned int v) {
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;
}

void radon_filter_sinogram_cuda(const float *x, float *y, const int batch_size, const int n_rays, const int n_angles) {
    const int rows = batch_size * n_angles;
    const int padded_size = next_power_of_two(n_rays * 2);
    // cuFFT only stores half of the coefficient because they are symmetric (see cuFFT documentation)
    const int fft_size = padded_size / 2 + 1;
    std::cout << "rows " << rows << std::endl;
    std::cout << "padded_size " << padded_size << std::endl;
    std::cout << "fft_size " << fft_size << std::endl;

    // pad x
    hipfftReal *padded_data = nullptr;
    checkCudaErrors(hipMalloc((void **) &padded_data, sizeof(hipfftReal) * rows * padded_size));
    checkCudaErrors(hipMemset(padded_data, 0, sizeof(hipfftReal) * rows * padded_size));
    checkCudaErrors(hipMemcpy2D(padded_data, sizeof(hipfftReal) * padded_size, x, sizeof(float) * n_rays,
                                 sizeof(float) * n_rays, rows, hipMemcpyDeviceToDevice));

    // allocate complex tensor to store FFT coefficients
    hipfftComplex *complex_data = nullptr;
    checkCudaErrors(hipMalloc((void **) &complex_data, sizeof(hipfftComplex) * rows * fft_size));

    // allocate real tensor to store padded filtered sinogram
    hipfftReal *filtered_padded_sino = nullptr;
    checkCudaErrors(hipMalloc((void **) &filtered_padded_sino, sizeof(hipfftReal) * rows * padded_size));
    checkCudaErrors(hipMemset(filtered_padded_sino, 0, sizeof(hipfftReal) * rows * padded_size));

    // create plans for FFT and iFFT
    hipfftHandle forward_plan, back_plan;
    checkCudaErrors(hipfftPlan1d(&forward_plan, padded_size, HIPFFT_R2C, rows));
    checkCudaErrors(hipfftPlan1d(&back_plan, padded_size, HIPFFT_C2R, rows));

    // do FFT
    checkCudaErrors(hipfftExecR2C(forward_plan, padded_data, complex_data));

    // filter in Fourier domain
    apply_filter << < dim3(fft_size / 16 + 1, rows / 16), dim3(16, 16) >> > (complex_data, fft_size);

    // do iFFT
    checkCudaErrors(hipfftExecC2R(back_plan, complex_data, filtered_padded_sino));

    // copy unpadded result in y
    checkCudaErrors(hipMemcpy2D(y, sizeof(float) * n_rays, filtered_padded_sino, sizeof(float) * padded_size,
                                 sizeof(float) * n_rays, rows, hipMemcpyDeviceToDevice));

    checkCudaErrors(hipfftDestroy(forward_plan));
    checkCudaErrors(hipfftDestroy(back_plan));
}
*/
