#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <hiprand/hiprand_kernel.h>
//#include <hiprand.h>
#include <hip/hip_fp16.h>


#include "utils.h"
#include "texture.h"

template<bool extend, int wpt>
__global__ void
radon_backward_kernel(float *output, hipTextureObject_t texture, const float *rays, const float *angles,
                      const int img_size, const int n_rays, const int n_angles) {

    __shared__ float s_sin[512];
    __shared__ float s_cos[512];

    // Calculate image coordinates
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;
    const uint batch_id = blockIdx.z * wpt;
    const uint tid = threadIdx.y * blockDim.x + threadIdx.x;

    for (int i = tid; i < n_angles; i += 256) {
        s_sin[i] = __sinf(angles[i]);
        s_cos[i] = __cosf(angles[i]);
    }
    __syncthreads();

    const float center = (img_size) / 2;
    const float max_r = center;
    float dx = (float) x - center + 0.5;
    float dy = (float) y - center + 0.5;

    float tmp[wpt];
#pragma unroll
    for (int i = 0; i < wpt; i++) tmp[i] = 0.0f;

    if (extend) {
//        if (r > max_r) {
//            dx *= max_r / r;
//            dy *= max_r / r;
//        }

        for (int i = 0; i < n_angles; i++) {
            float j = s_cos[i] * dx + s_sin[i] * dy + center;
#pragma unroll
            for (int b = 0; b < wpt; b++) {
                tmp[b] += tex2DLayered<float>(texture, j, i + 0.5f, batch_id + b);
            }
        }
    } else {
        const float r = hypot(dx, dy);
        if (r <= max_r) {
            for (int i = 0; i < n_angles; i++) {
                float j = s_cos[i] * dx + s_sin[i] * dy + center;
#pragma unroll
                for (int b = 0; b < wpt; b++) {
                    tmp[b] += tex2DLayered<float>(texture, j, i + 0.5f, batch_id + b);
                }
            }
        }
    }

#pragma unroll
    for (int b = 0; b < wpt; b++) {
        output[(batch_id + b) * img_size * img_size + y * img_size + x] = tmp[b];
    }
}

void radon_backward_cuda(const float *x, const float *rays, const float *angles, float *y, TextureCache &tex_cache,
                         const int batch_size, const int img_size, const int n_rays, const int n_angles,
                         const int device, const bool extend) {
    // copy x into CUDA Array (allocating it if needed) and bind to texture
    Texture *tex = tex_cache.get({device, batch_size, n_rays, n_angles, 1, PRECISION_FLOAT});
    tex->put(x);

    // if batch size is multiple of 4 each thread does 4 batches (is faster) (wpt = work per thread)
    const int wpt = (batch_size % 4 == 0) ? 4 : 1;
    const int grid_size = img_size / 16;
    dim3 dimGrid(grid_size, grid_size, batch_size / wpt);
    dim3 dimBlock(16, 16);

    // Invoke kernel
    if (extend) {
        if (wpt == 4) {
            radon_backward_kernel<true, 4> << < dimGrid, dimBlock >> >
                                                         (y, tex->texture, rays, angles, img_size, n_rays, n_angles);
        } else {
            radon_backward_kernel<true, 1> << < dimGrid, dimBlock >> >
                                                         (y, tex->texture, rays, angles, img_size, n_rays, n_angles);
        }
    } else {
        if (wpt == 4) {
            radon_backward_kernel<false, 4> << < dimGrid, dimBlock >> >
                                                          (y, tex->texture, rays, angles, img_size, n_rays, n_angles);
        } else {
            radon_backward_kernel<false, 1> << < dimGrid, dimBlock >> >
                                                          (y, tex->texture, rays, angles, img_size, n_rays, n_angles);
        }
    }
}


template<bool extend>
__global__ void
radon_backward_kernel_half(__half *output, hipTextureObject_t texture, const float *rays, const float *angles,
                           const int img_size, const int n_rays, const int n_angles) {

    __shared__ float s_sin[512];
    __shared__ float s_cos[512];

    // Calculate image coordinates
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;
    const uint batch_id = blockIdx.z * 4;
    const uint tid = threadIdx.y * blockDim.x + threadIdx.x;

    for (int i = tid; i < n_angles; i += 256) {
        s_sin[i] = __sinf(angles[i]);
        s_cos[i] = __cosf(angles[i]);
    }
    __syncthreads();

    const float center = (img_size) / 2;
    const float max_r = center;
    float dx = (float) x - center + 0.5;
    float dy = (float) y - center + 0.5;

    float tmp[4];
#pragma unroll
    for (int i = 0; i < 4; i++) tmp[i] = 0.0f;

    if (extend) {
        for (int i = 0; i < n_angles; i++) {
            float j = s_cos[i] * dx + s_sin[i] * dy + center;

            // read 4 values at the given position and accumulate
            float4 read = tex2DLayered<float4>(texture, j, i + 0.5f, blockIdx.z);
            tmp[0] += read.x;
            tmp[1] += read.y;
            tmp[2] += read.z;
            tmp[3] += read.w;
        }
    } else {
        const float r = hypot(dx, dy);
        if (r <= max_r) {
            for (int i = 0; i < n_angles; i++) {
                float j = s_cos[i] * dx + s_sin[i] * dy + center;

                // read 4 values at the given position and accumulate
                float4 read = tex2DLayered<float4>(texture, j, i + 0.5f, blockIdx.z);
                tmp[0] += read.x;
                tmp[1] += read.y;
                tmp[2] += read.z;
                tmp[3] += read.w;
            }
        }
    }

#pragma unroll
    for (int b = 0; b < 4; b++) {
        output[(batch_id + b) * img_size * img_size + y * img_size + x] = __float2half(tmp[b]);
    }
}

void radon_backward_cuda(const unsigned short *x, const float *rays, const float *angles, unsigned short *y,
                         TextureCache &tex_cache,
                         const int batch_size, const int img_size, const int n_rays, const int n_angles,
                         const int device, const bool extend) {
    // copy x into CUDA Array (allocating it if needed) and bind to texture
    Texture *tex = tex_cache.get({device, batch_size, n_rays, n_angles, 4, PRECISION_HALF});
    tex->put(x);

    const int grid_size = img_size / 16;
    dim3 dimGrid(grid_size, grid_size, batch_size / 4);
    dim3 dimBlock(16, 16);

    // Invoke kernel
    if (extend) {
        radon_backward_kernel_half<true> << < dimGrid, dimBlock >> >
                                                       ((__half *) y, tex->texture, rays, angles, img_size, n_rays, n_angles);
    } else {
        radon_backward_kernel_half<false> << < dimGrid, dimBlock >> >
                                                        ((__half *) y, tex->texture, rays, angles, img_size, n_rays, n_angles);
    }
}


/*
template<typename T> __host__ __device__

inline T lerp(T v0, T v1, T t) {
    return fma(t, v1, fma(-t, v0, v0));
}

template<bool extend, int wpt, int threads>
__global__ void radon_backward_kernel_lb(float *output, const float *sinogram, const float *rays, const float *angles,
                                         const int img_size, const int n_rays, const int n_angles,
                                         const int batch_size) {

    __shared__ float s_sin[512];
    __shared__ float s_cos[512];

    // Calculate image coordinates
    const uint batch_id = blockIdx.x * blockDim.x * wpt + threadIdx.x;
    const uint x = blockIdx.y * blockDim.y + threadIdx.y;
    const uint y = blockIdx.z;

    const uint tid = threadIdx.y * blockDim.x + threadIdx.x;

    for (int i = tid; i < n_angles; i += 256) {
        s_sin[i] = __sinf(angles[i]);
        s_cos[i] = __cosf(angles[i]);
    }
    __syncthreads();

    const float center = img_size / 2.0f - 0.5f;
    const float max_r = center;
    float dx = (float) x - center;
    float dy = (float) y - center;

    float tmp[wpt];
    for(int i = 0; i < wpt;i++) tmp[i] = 0.0f;
    const float r = hypot(dx, dy);

    for (int i = 0; i < n_angles; i++) {
        float j = s_cos[i] * dx + s_sin[i] * dy + center;
        float j_ceil = ceilf(j);
        float j_floor = j_ceil - 1.0f;
        float t = j - j_floor;
        const int base = i * img_size * batch_size + int(j_floor) * batch_size;
        if(j_floor >= 0 && j_ceil < img_size) {
            for(int b = 0; b < wpt; b++) {
                tmp[b] += lerp(sinogram[base + b*threads + batch_id],
                            sinogram[base + batch_size + b*threads + batch_id], t);

            }
        }
//        else{
//            if(j_floor < 0 && j_ceil >= 0) tmp[0] += lerp(0.0f, sinogram[base + batch_size + batch_id], t);
//            if(j_ceil >= img_size && j_floor < img_size) tmp[0] += lerp(sinogram[base + batch_id], 0.0f, t);
//        }
    }

    for(int b = 0; b < wpt; b++) {
        output[(batch_id + b*threads) * img_size * img_size + y * img_size + x] = tmp[b];
    }
}

void radon_backward_cuda_lb(const float *x, const float *rays, const float *angles, float *y, TextureCache &tex_cache,
                            const int batch_size, const int img_size, const int n_rays, const int n_angles,
                            const int device, const bool extend) {
    checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(radon_backward_kernel_lb<true), 4, 32>, hipFuncCachePreferL1));

    // Invoke kernel
    const int grid_size = img_size / 1;
    dim3 dimGrid(batch_size / (256), grid_size, img_size);
    dim3 dimBlock(256, 1);

    radon_backward_kernel_lb<true, 1, 128> << < dimGrid, dimBlock >> >
                                                        (y, x, rays, angles, img_size, n_rays, n_angles, batch_size);
//    if (extend) {
//        radon_backward_kernel_lb<true, 4, 32> << < dimGrid, dimBlock >> >
//                                                     (y, x, rays, angles, img_size, n_rays, n_angles, batch_size);
//    } else {
//        radon_backward_kernel_lb<false, 1, 32> << < dimGrid, dimBlock >> >
//                                                      (y, x, rays, angles, img_size, n_rays, n_angles, batch_size);
//    }
}
*/