#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#include "utils.h"
#include "texture.h"

template < bool extend, int wpt >
__global__ void radon_backward_kernel(float *output, hipTextureObject_t texObj, const float *rays, const float *angles,
                                      const int img_size, const int n_rays, const int n_angles) {

    __shared__ float s_sin[512];
    __shared__ float s_cos[512];

    // Calculate image coordinates
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;
    const uint batch_id = blockIdx.z * wpt;
    const uint tid = threadIdx.y * blockDim.x + threadIdx.x;

    for (int i = tid; i < n_angles; i += 256) {
        s_sin[i] = __sinf(angles[i]);
        s_cos[i] = __cosf(angles[i]);
    }
    __syncthreads();

    const float center = (img_size) / 2;
    const float max_r = center;
    float dx = (float) x - center + 0.5;
    float dy = (float) y - center + 0.5;

    float tmp[wpt];
#pragma unroll
    for(int i = 0; i < wpt;i++) tmp[i] = 0.0f;

    if (extend) {
//        if (r > max_r) {
//            dx *= max_r / r;
//            dy *= max_r / r;
//        }

        for (int i = 0; i < n_angles; i++) {
            float j = s_cos[i] * dx + s_sin[i] * dy + center;
#pragma unroll
            for(int b = 0; b < wpt; b++) {
                tmp[b] += tex2DLayered<float>(texObj, j, i + 0.5f, batch_id + b);
            }
        }
    }
    else {
        const float r = hypot(dx, dy);
        if (r <= max_r) {
            for (int i = 0; i < n_angles; i++) {
                float j = s_cos[i] * dx + s_sin[i] * dy + center;
#pragma unroll
                for(int b = 0; b < wpt; b++) {
                    tmp[b] += tex2DLayered<float>(texObj, j, i + 0.5f, batch_id + b);
                }            }
        }
    }

#pragma unroll
    for(int b = 0; b < wpt; b++) {
        output[(batch_id+b) * img_size * img_size + y * img_size + x] = tmp[b];
    }
}

void radon_backward_cuda(const float *x, const float *rays, const float *angles, float *y, TextureCache &tex_cache,
                         const int batch_size, const int img_size, const int n_rays, const int n_angles,
                         const int device, const bool extend) {
    // copy x into CUDA Array (allocating it if needed) and bind to texture
    Texture *tex = tex_cache.get({device, batch_size, n_rays, n_angles});
    tex->put(x);

    // Invoke kernel
    const int wpt = (batch_size % 4 == 0)? 4 : 1;
    const int grid_size = img_size / 16;
    dim3 dimGrid(grid_size, grid_size, batch_size / wpt);
    dim3 dimBlock(16, 16);

    if (extend) {
        if(wpt == 4){
            radon_backward_kernel<true, 4> << < dimGrid, dimBlock >> >
                                                           (y, tex->texObj, rays, angles, img_size, n_rays, n_angles);
        }else{
            radon_backward_kernel<true, 1> << < dimGrid, dimBlock >> >
                                                           (y, tex->texObj, rays, angles, img_size, n_rays, n_angles);
        }
    }
    else {
        if(wpt == 4){
            radon_backward_kernel<false, 4> << < dimGrid, dimBlock >> >
                                                         (y, tex->texObj, rays, angles, img_size, n_rays, n_angles);
        }else{
            radon_backward_kernel<false, 1> << < dimGrid, dimBlock >> >
                                                         (y, tex->texObj, rays, angles, img_size, n_rays, n_angles);
        }
    }
}

template<typename T> __host__ __device__

inline T lerp(T v0, T v1, T t) {
    return fma(t, v1, fma(-t, v0, v0));
}

template<bool extend, int wpt, int threads>
__global__ void radon_backward_kernel_lb(float *output, const float *sinogram, const float *rays, const float *angles,
                                         const int img_size, const int n_rays, const int n_angles,
                                         const int batch_size) {

    __shared__ float s_sin[512];
    __shared__ float s_cos[512];

    // Calculate image coordinates
    const uint batch_id = blockIdx.x * blockDim.x * wpt + threadIdx.x;
    const uint x = blockIdx.y * blockDim.y + threadIdx.y;
    const uint y = blockIdx.z;

    const uint tid = threadIdx.y * blockDim.x + threadIdx.x;

    for (int i = tid; i < n_angles; i += 256) {
        s_sin[i] = __sinf(angles[i]);
        s_cos[i] = __cosf(angles[i]);
    }
    __syncthreads();

    const float center = img_size / 2.0f - 0.5f;
    const float max_r = center;
    float dx = (float) x - center;
    float dy = (float) y - center;

    float tmp[wpt];
    for(int i = 0; i < wpt;i++) tmp[i] = 0.0f;
    const float r = hypot(dx, dy);

    for (int i = 0; i < n_angles; i++) {
        float j = s_cos[i] * dx + s_sin[i] * dy + center;
        float j_ceil = ceilf(j);
        float j_floor = j_ceil - 1.0f;
        float t = j - j_floor;
        const int base = i * img_size * batch_size + int(j_floor) * batch_size;
        if(j_floor >= 0 && j_ceil < img_size) {
            for(int b = 0; b < wpt; b++) {
                tmp[b] += lerp(sinogram[base + b*threads + batch_id],
                            sinogram[base + batch_size + b*threads + batch_id], t);

            }
        }
//        else{
//            if(j_floor < 0 && j_ceil >= 0) tmp[0] += lerp(0.0f, sinogram[base + batch_size + batch_id], t);
//            if(j_ceil >= img_size && j_floor < img_size) tmp[0] += lerp(sinogram[base + batch_id], 0.0f, t);
//        }
    }

    for(int b = 0; b < wpt; b++) {
        output[(batch_id + b*threads) * img_size * img_size + y * img_size + x] = tmp[b];
    }
}

void radon_backward_cuda_lb(const float *x, const float *rays, const float *angles, float *y, TextureCache &tex_cache,
                            const int batch_size, const int img_size, const int n_rays, const int n_angles,
                            const int device, const bool extend) {
    checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(radon_backward_kernel_lb<true), 4, 32>, hipFuncCachePreferL1));

    // Invoke kernel
    const int grid_size = img_size / 1;
    dim3 dimGrid(batch_size / (256), grid_size, img_size);
    dim3 dimBlock(256, 1);

    radon_backward_kernel_lb<true, 1, 128> << < dimGrid, dimBlock >> >
                                                        (y, x, rays, angles, img_size, n_rays, n_angles, batch_size);
//    if (extend) {
//        radon_backward_kernel_lb<true, 4, 32> << < dimGrid, dimBlock >> >
//                                                     (y, x, rays, angles, img_size, n_rays, n_angles, batch_size);
//    } else {
//        radon_backward_kernel_lb<false, 1, 32> << < dimGrid, dimBlock >> >
//                                                      (y, x, rays, angles, img_size, n_rays, n_angles, batch_size);
//    }
}
