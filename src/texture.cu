#include "texture.h"
#include <iostream>

Texture::Texture(DeviceSizeKey k) : key(k) {
    checkCudaErrors(hipSetDevice(this->key.device));

#ifdef VERBOSE
    std::cout << "[TORCH RADON] Allocating Texture " << this->key << std::endl;
#endif

    // Allocate a layered CUDA array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    const hipExtent extent = make_hipExtent(k.width, k.height, k.batch);
    checkCudaErrors(hipMalloc3DArray(&array, &channelDesc, extent, hipArrayLayered));

    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = array;

    // Specify texture object parameters
    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeBorder;
    texDesc.addressMode[1] = hipAddressModeBorder;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // Create texture object
    checkCudaErrors(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL));
}

void Texture::put(const float *data) {
    checkCudaErrors(hipSetDevice(this->key.device));
    const uint pitch = this->key.width;

    // copy data into array
    hipMemcpy3DParms myparms = {0};
    myparms.srcPos = make_hipPos(0, 0, 0);
    myparms.dstPos = make_hipPos(0, 0, 0);
    myparms.srcPtr = make_hipPitchedPtr((void *) data, pitch * sizeof(float), this->key.width, this->key.height);
    myparms.dstArray = this->array;
    myparms.extent = make_hipExtent(this->key.width, this->key.height, this->key.batch);
    myparms.kind = hipMemcpyDeviceToDevice;
    checkCudaErrors(hipMemcpy3D(&myparms));
}

bool Texture::matches(DeviceSizeKey& k){
    return k == this->key;
}

Texture::~Texture() {
#ifdef VERBOSE
    std::cout << "[TORCH RADON] Freeing Texture " << this->key << std::endl;
#endif
    if (this->array != nullptr) {
        checkCudaErrors(hipSetDevice(this->key.device));
        checkCudaErrors(hipFreeArray(this->array));
        checkCudaErrors(hipDestroyTextureObject(this->texObj));
        this->array = nullptr;
    }
}