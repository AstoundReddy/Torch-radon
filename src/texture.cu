#include "hip/hip_runtime.h"
#include "texture.h"
#include <iostream>
#include <hip/hip_fp16.h>


hipChannelFormatDesc get_channel_desc(int channels, int precision) {
    if (precision == PRECISION_FLOAT) {
        return hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    }
    if (precision == PRECISION_HALF && channels == 4) {
        return hipCreateChannelDesc(16, 16, 16, 16, hipChannelFormatKindFloat);
    }
    std::cerr << "[TORCH RADON] ERROR unsupported number of channels and precision (channels:" << channels
              << ", precision: " << precision << ")" << std::endl;
    return hipCreateChannelDesc(16, 16, 16, 16, hipChannelFormatKindFloat);
}

Texture::Texture(DeviceSizeKey k) : key(k) {
    checkCudaErrors(hipSetDevice(this->key.device));

#ifdef VERBOSE
    std::cout << "[TORCH RADON] Allocating Texture " << this->key << std::endl;
#endif

    // Allocate a layered CUDA array
    hipChannelFormatDesc channelDesc = get_channel_desc(key.channels, key.precision);
    const hipExtent extent = make_hipExtent(k.width, k.height, k.batch / key.channels);
    checkCudaErrors(hipMalloc3DArray(&array, &channelDesc, extent, hipArrayLayered));

    // Create resource descriptor
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = array;

    // Specify texture object parameters
    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeBorder;
    texDesc.addressMode[1] = hipAddressModeBorder;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // Create texture object
    checkCudaErrors(hipCreateTextureObject(&texture, &resDesc, &texDesc, NULL));

    // Create surface object
    checkCudaErrors(hipCreateSurfaceObject(&surface, &resDesc));
}

void Texture::put(const float *data) {
    if (this->key.precision == PRECISION_HALF) {
        std::cerr << "[TORCH RADON] ERROR putting half precision data into a float texture" << std::endl;
    }

    checkCudaErrors(hipSetDevice(this->key.device));
    const uint pitch = this->key.width;

    // copy data into array
    hipMemcpy3DParms myparms = {0};
    myparms.srcPos = make_hipPos(0, 0, 0);
    myparms.dstPos = make_hipPos(0, 0, 0);
    myparms.srcPtr = make_hipPitchedPtr((void *) data, pitch * sizeof(float), this->key.width, this->key.height);
    myparms.dstArray = this->array;
    myparms.extent = make_hipExtent(this->key.width, this->key.height, this->key.batch);
    myparms.kind = hipMemcpyDeviceToDevice;
    checkCudaErrors(hipMemcpy3D(&myparms));
}

__global__ void
write_half_to_surface(const __half *data, hipSurfaceObject_t surface, const int b_stride, const int h_stride) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int b = 4 * blockIdx.z;

    __half tmp[4];
    for (int i = 0; i < 4; i++) tmp[i] = __float2half(data[(b + i) * b_stride + y * h_stride + x]);

    surf2DLayeredwrite<float2>(*(float2 *) tmp, surface, x * sizeof(float2), y, blockIdx.z);
}

void Texture::put(const unsigned short *data) {
    if (this->key.precision == PRECISION_FLOAT) {
        std::cerr << "[TORCH RADON] ERROR putting single precision data into a half precision texture" << std::endl;
    }

    checkCudaErrors(hipSetDevice(this->key.device));

    write_half_to_surface << < dim3(key.width / 16, key.height / 16, key.batch / 4), dim3(16, 16) >> >
                                                                                     ((__half *) data, this->surface,
                                                                                             key.width *
                                                                                             key.height, key.width);

    checkCudaErrors(hipDeviceSynchronize());
}

bool Texture::matches(DeviceSizeKey &k) {
    return k == this->key;
}

Texture::~Texture() {
#ifdef VERBOSE
    std::cout << "[TORCH RADON] Freeing Texture " << this->key << std::endl;
#endif
    if (this->array != nullptr) {
        checkCudaErrors(hipSetDevice(this->key.device));
        checkCudaErrors(hipFreeArray(this->array));
        checkCudaErrors(hipDestroyTextureObject(this->texture));
        checkCudaErrors(hipDestroySurfaceObject(this->surface));
        this->array = nullptr;
    }
}