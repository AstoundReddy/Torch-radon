#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

static texture<float, 2> gT_FanVolumeTexture;


static bool bindVolumeDataTexture(float* data, hipArray*& dataArray, unsigned int pitch, unsigned int width, unsigned int height)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	dataArray = 0;
	hipMallocArray(&dataArray, &channelDesc, width, height);
	hipMemcpy2DToArray(dataArray, 0, 0, data, pitch*sizeof(float), width*sizeof(float), height, hipMemcpyDeviceToDevice);

	gT_FanVolumeTexture.addressMode[0] = hipAddressModeBorder;
	gT_FanVolumeTexture.addressMode[1] = hipAddressModeBorder;
	gT_FanVolumeTexture.filterMode = hipFilterModeLinear;
	gT_FanVolumeTexture.normalized = false;

	// TODO: For very small sizes (roughly <=512x128) with few angles (<=180)
	// not using an array is more efficient.
	//hipBindTexture2D(0, gT_FanVolumeTexture, (const void*)data, channelDesc, width, height, sizeof(float)*pitch);
	hipBindTextureToArray(gT_FanVolumeTexture, dataArray, channelDesc);

	// TODO: error value?

	return true;
}
