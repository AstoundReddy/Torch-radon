#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

texture<float, 2> bindVolumeDataTexture(float* data, hipArray*& dataArray, unsigned int pitch, unsigned int width, unsigned int height)
{
    texture<float, 2> my_tex;

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	dataArray = 0;
	hipMallocArray(&dataArray, &channelDesc, width, height);
	hipMemcpy2DToArray(dataArray, 0, 0, data, pitch*sizeof(float), width*sizeof(float), height, hipMemcpyDeviceToDevice);

	my_tex.addressMode[0] = hipAddressModeBorder;
	my_tex.addressMode[1] = hipAddressModeBorder;
	my_tex.filterMode = hipFilterModeLinear;
	my_tex.normalized = false;

	// TODO: For very small sizes (roughly <=512x128) with few angles (<=180)
	// not using an array is more efficient.
	//hipBindTexture2D(0, gT_FanVolumeTexture, (const void*)data, channelDesc, width, height, sizeof(float)*pitch);
	hipBindTextureToArray(gT_FanVolumeTexture, dataArray, channelDesc);

	// TODO: error value?

	return my_tex;
}

__global__ void transformKernel(float* output, texture<float, 2> texObj) {
    // Calculate texture coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Read from texture and write to global memory
    output[y * 128 + x] = tex2D(texObj, x+0.5, y+0.5);
}

torch::Tensor copy_image_cuda(torch::Tensor x){
    hipArray* tmp;
    auto my_tex = bindVolumeDataTexture(x.data<float>(), tmp, 128, 128, 128);

    auto y = torch::zeros_like(x);

    // Invoke kernel
    dim3 dimGrid(8, 8);
    dim3 dimBlock(16, 16);

    transformKernel<<<dimGrid, dimBlock>>>(y.data<float>(), my_tex);

    return y;
}
