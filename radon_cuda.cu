#include "hip/hip_runtime.h"
#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "utils.h"

__global__ void radon_forward_kernel(float* output, hipTextureObject_t texObj, const float* rays, const float* angles, const int img_size, const int n_rays, const int n_angles) {
    // Calculate texture coordinates
    const uint ray_id = blockIdx.x * blockDim.x + threadIdx.x;
    const uint batch_id = blockIdx.y;
    const float rsx = rays[ray_id*4+0];
    const float rsy = rays[ray_id*4+1];
    const float rex = rays[ray_id*4+2];
    const float rey = rays[ray_id*4+3];
    const float v = img_size/2;

    const float vx = (rex-rsx)/img_size;
    const float vy = (rey-rsy)/img_size;
    const float n = hypot(vx, vy);

    for(int i = 0; i < n_angles; i++){
        // rotate ray
        float angle = angles[i];
        float cs = __cosf(angle);
        float sn = __sinf(angle);

        float sx = rsx*cs - rsy*sn + v;
        float sy = rsx*sn + rsy*cs + v;
        float rvx = vx*cs - vy*sn;
        float rvy = vx*sn + vy*cs;

        float tmp = 0.0;
        for(int j = 0; j < img_size; j++){
            tmp += tex2DLayered<float>(texObj, sx+rvx*j, sy+rvy*j, batch_id);
        }

        output[batch_id*n_rays*n_angles + i*n_rays + ray_id] = tmp*n;
    }
}

void radon_forward_cuda(const float* x, const float* rays, const float* angles, float* y, const int batch_size, const int img_size, const int n_rays, const int n_angles){
    hipArray* tmp;
    auto my_tex = create_texture(x, tmp, batch_size, img_size, img_size, img_size);

    // Invoke kernel
    dim3 dimGrid(8, batch_size);
    dim3 dimBlock(16);

    radon_forward_kernel<<<dimGrid, dimBlock>>>(y, my_tex, rays, angles, img_size, n_rays, n_angles);

    hipFreeArray(tmp);
}

__global__ void radon_backward_kernel(float* output, hipTextureObject_t texObj, const float* rays, const float* angles, const int img_size, const int n_rays, const int n_angles) {
    // Calculate texture coordinates
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;
    const uint batch_id = blockIdx.z;
    const float dx = (float)x - img_size/2  + 0.5;
    const float dy = (float)y - img_size/2  + 0.5;
    const float v = img_size/2;
    float tmp = 0.0;

    for(int i = 0; i < n_angles; i++){
        // TODO cache angles
        float angle = angles[i];
        float j = __cosf(angle) * dx + __sinf(angle) * dy + v;
        tmp += tex2DLayered<float>(texObj, j, i+0.5f, batch_id);
    }

    output[batch_id*img_size*img_size + y*img_size + x] = tmp;
}

void radon_backward_cuda(const float* x, const float* rays, const float* angles, float* y, const int batch_size, const int img_size, const int n_rays, const int n_angles){
    //std::cout << batch_size << " " << img_size << " " << n_rays << " " << n_angles << std::endl;
    hipArray* tmp;
    auto my_tex = create_texture(x, tmp, batch_size, n_rays, n_angles, n_rays);

    // Invoke kernel
    dim3 dimGrid(8, 8, batch_size);
    dim3 dimBlock(16, 16);

    radon_backward_kernel<<<dimGrid, dimBlock>>>(y, my_tex, rays, angles, img_size, n_rays, n_angles);

    hipFreeArray(tmp);
}


#define MIN(a,b) (((a)<(b))?(a):(b))
__global__ void apply_filter(hipfftComplex* sino,const float* f) {
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;

    const int numSensors = 256;
    const float k = MIN(float(x), 256.0f - float(x)) / (float)numSensors;

    sino[numSensors*y + x].x *= k;
    sino[numSensors*y + x].y *= k;
}


void radon_filter_sinogram_cuda(const float* x, const float* f, float* y, const int batch_size, const int n_rays, const int n_angles){
    //TODO int padded_size = next_power_of_two(n_rays);
    int padded_size = n_rays*2;
    hipfftReal* padded_data = nullptr;// (hipfftReal*) x;
    checkCudaErrors(hipMalloc((void**)&padded_data, sizeof(hipfftReal)*batch_size*n_angles*padded_size));
    checkCudaErrors(hipMemcpy2D(padded_data, sizeof(hipfftReal)*padded_size, x, sizeof(float)*n_rays, sizeof(float)*n_rays, batch_size*n_angles, hipMemcpyDeviceToDevice));


    hipfftComplex* complex_data = nullptr;
    checkCudaErrors(hipMalloc((void**)&complex_data, sizeof(hipfftComplex)*batch_size*n_angles*padded_size));

    hipfftHandle forward_plan, back_plan;
    checkCudaErrors(hipfftPlan1d(&forward_plan, padded_size, HIPFFT_R2C, batch_size*n_angles));
    checkCudaErrors(hipfftPlan1d(&back_plan, padded_size, HIPFFT_C2R, batch_size*n_angles));

    checkCudaErrors(hipfftExecR2C(forward_plan, padded_data, complex_data));

    // TODO è fatto a cazzo
    apply_filter<<<dim3(16, n_angles), dim3(16, 16)>>>(complex_data, f);

    // TODO pad also y
    checkCudaErrors(hipfftExecC2R(back_plan, complex_data, y));
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipfftDestroy(forward_plan));
    checkCudaErrors(hipfftDestroy(back_plan));
}

int main(){
    std::cout << "float: " << sizeof(float) << std::endl;
    std::cout << "hipfftReal: " << sizeof(hipfftReal) << std::endl;
    std::cout << "hipfftComplex: " << sizeof(hipfftComplex) << std::endl;
    std::cout << "Hello CUDA" << std::endl;
}
