#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "utils.h"

__global__ void radon_forward_kernel(float* output, hipTextureObject_t texObj, const float* rays, const float* angles, const int img_size, const int n_rays, const int n_angles) {
    // Calculate texture coordinates
    const uint ray_id = blockIdx.x * blockDim.x + threadIdx.x;
    const uint batch_id = blockIdx.y;
    const float rsx = rays[ray_id*4+0];
    const float rsy = rays[ray_id*4+1];
    const float rex = rays[ray_id*4+2];
    const float rey = rays[ray_id*4+3];

    for(int i = 0; i < n_angles; i++){
        // rotate ray
        float angle = angles[i];
        float sx = rsx*cos(angle) - rsy*sin(angle) + img_size/2;
        float sy = rsx*sin(angle) + rsy*cos(angle) + img_size/2;
        float ex = rex*cos(angle) - rey*sin(angle) + img_size/2;
        float ey = rex*sin(angle) + rey*cos(angle) + img_size/2;

        float vx = (ex-sx)/img_size;
        float vy = (ey-sy)/img_size;
        float n = hypot(vx, vy);

        float tmp = 0.0;
        for(int j = 0; j < img_size; j++){
            tmp += tex2DLayered<float>(texObj, sx+vx*j, sy+vy*j, batch_id);
        }

        output[batch_id*n_rays*n_angles + i*n_rays + ray_id] = tmp*n;
    }
}

void radon_forward_cuda(const float* x, const float* rays, const float* angles, float* y, const int batch_size, const int img_size, const int n_rays, const int n_angles){
    hipArray* tmp;
    auto my_tex = create_texture(x, tmp, batch_size, img_size, img_size, img_size);

    // Invoke kernel
    dim3 dimGrid(8, batch_size);
    dim3 dimBlock(16);

    radon_forward_kernel<<<dimGrid, dimBlock>>>(y, my_tex, rays, angles, img_size, n_rays, n_angles);

    hipFreeArray(tmp);
}

__global__ void radon_backward_kernel(float* output, hipTextureObject_t texObj, const float* rays, const float* angles, const int img_size, const int n_rays, const int n_angles) {
    // Calculate texture coordinates
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;
    const uint batch_id = blockIdx.z;

    float tmp = 0.0;

    for(int i = 0; i < n_angles; i++){
        float angle = angles[i];
        float j = cos(angle) * ((float)x - img_size/2  + 0.5) + sin(angle) * ((float)y - img_size/2  + 0.5) + img_size/2;

        tmp += tex2DLayered<float>(texObj, j, i+0.5, batch_id);
    }

    output[batch_id*img_size*img_size + y*img_size + x] = tmp;
}

void radon_backward_cuda(const float* x, const float* rays, const float* angles, float* y, const int batch_size, const int img_size, const int n_rays, const int n_angles){
    //std::cout << batch_size << " " << img_size << " " << n_rays << " " << n_angles << std::endl;
    hipArray* tmp;
    auto my_tex = create_texture(x, tmp, batch_size, n_rays, n_angles, n_rays);

    // Invoke kernel
    dim3 dimGrid(8, 8, batch_size);
    dim3 dimBlock(16, 16);

    radon_backward_kernel<<<dimGrid, dimBlock>>>(y, my_tex, rays, angles, img_size, n_rays, n_angles);

    hipFreeArray(tmp);
}


int main(){
    std::cout << "Hello CUDA" << std::endl;
}
