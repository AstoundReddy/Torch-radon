#include "hip/hip_runtime.h"
//#include <torch/extension.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

texture<float, 2> bindVolumeDataTexture(float* data, hipArray*& dataArray, unsigned int pitch, unsigned int width, unsigned int height)
{
    texture<float, 2> my_tex;

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	dataArray = 0;
	hipMallocArray(&dataArray, &channelDesc, width, height);
	hipMemcpy2DToArray(dataArray, 0, 0, data, pitch*sizeof(float), width*sizeof(float), height, hipMemcpyDeviceToDevice);

	my_tex.addressMode[0] = hipAddressModeBorder;
	my_tex.addressMode[1] = hipAddressModeBorder;
	my_tex.filterMode = hipFilterModeLinear;
	my_tex.normalized = false;

	// TODO: For very small sizes (roughly <=512x128) with few angles (<=180)
	// not using an array is more efficient.
	//hipBindTexture2D(0, my_tex, (const void*)data, channelDesc, width, height, sizeof(float)*pitch);
	hipBindTextureToArray(my_tex, dataArray, channelDesc);

	// TODO: error value?

	return my_tex;
}

__global__ void transformKernel(float* output, hipTextureObject_t texObj) {
    // Calculate texture coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Read from texture and write to global memory
    output[y * 128 + x] = tex2D<float>(texObj, x+0.5, y+0.5);
}

hipTextureObject_t create_texture(float* data, unsigned int width, unsigned int height){
    unsigned int pitch = width;

    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc =
               hipCreateChannelDesc(32, 0, 0, 0,
                                     hipChannelFormatKindFloat);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, width, height);
    hipMemcpy2DToArray(cuArray, 0, 0, data, pitch*sizeof(float), width*sizeof(float), height, hipMemcpyDeviceToDevice);

    // Specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    // Specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeBorder;
    texDesc.addressMode[1]   = hipAddressModeBorder;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // Create texture object
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

    return texObj;
}

//torch::Tensor copy_image_cuda(torch::Tensor x){
void copy_image_cuda(float* x, float* y){
    //hipArray* tmp;
    //auto my_tex = bindVolumeDataTexture(x.data<float>(), tmp, 128, 128, 128);

    //auto y = torch::zeros_like(x);
    
    auto my_tex = create_texture(x, 128, 128);

    // Invoke kernel
    dim3 dimGrid(8, 8);
    dim3 dimBlock(16, 16);

    transformKernel<<<dimGrid, dimBlock>>>(y, my_tex);

    //return y;
}

int main(){
    std::cout << "Hello CUDA" << std::endl;
}